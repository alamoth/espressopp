#include "StorageGPU.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUERR { \
    hipError_t cudaerr; \
    if ((cudaerr = hipGetLastError()) != hipSuccess){ \
        printf("CUDA ERROR: \"%s\" in File %s at LINE %d.\n", hipGetErrorString(cudaerr), __FILE__, __LINE__); \
    } \
}

void gpu_resizeParticleData(    int N, 
                                int **d_cellId,
                                int **d_id,
                                int **d_type, 
                                realG **d_drift,
                                realG **d_mass, 
                                realG3 **d_pos,
                                realG3 **d_force,
                                bool **d_real
                            ){
    int numBytesD = N * sizeof(realG);
    int numBytesI = N * sizeof(int);

    if(*d_cellId != 0) hipFree(*d_cellId);                                                             CUERR
    if(*d_id != 0) hipFree(*d_id);                                                                     CUERR
    if(*d_type != 0) hipFree(*d_type);                                                                 CUERR
    if(*d_drift != 0) hipFree(*d_drift);                                                               CUERR
    if(*d_mass != 0) hipFree(*d_mass);                                                                 CUERR
    if(*d_pos != 0) hipFree(*d_pos);                                                                   CUERR
    if(*d_force != 0) hipFree(*d_force);                                                               CUERR
    if(*d_real != 0) hipFree(*d_real);                                                                 CUERR

    hipMalloc(d_cellId, numBytesI);                                                                    CUERR 
    hipMalloc(d_id, numBytesI);                                                                        CUERR 
    hipMalloc(d_type, numBytesI);                                                                      CUERR
    hipMalloc(d_pos,   sizeof(realG3) * N);                                                           CUERR
    hipMalloc(d_force, sizeof(realG3) * N);                                                           CUERR
    hipMalloc(d_mass, numBytesD);                                                                      CUERR
    hipMalloc(d_drift, numBytesD);                                                                     CUERR
    hipMalloc(d_real, sizeof(bool) * N);                                                               CUERR

    hipMemset(*d_force, 0, sizeof(realG3) * N);                                                       CUERR
    
}

void gpu_h2dCellData(   int M, 
                        int *h_cellNeighbors,
                        int **d_cellNeighbors
                    ) {
    int numBytesI = M * sizeof(int);
    hipMemcpy(*d_cellNeighbors, h_cellNeighbors, numBytesI * 27, hipMemcpyHostToDevice);              CUERR
}

void gpu_resizeCellData(    int M,
                            int **d_cellOffsets,
                            int **d_particlesCell,
                            int **d_cellNeighbors) {

    int numBytes = M * sizeof(int);

    if(*d_cellOffsets != 0 && *d_particlesCell != 0){
        hipFree(*d_cellOffsets);                                                                       CUERR
        hipFree(*d_particlesCell);                                                                     CUERR
        hipFree(*d_cellNeighbors);                                                                     CUERR
    }

    hipMalloc(d_cellOffsets, numBytes);                                                                CUERR
    hipMalloc(d_particlesCell, numBytes);                                                              CUERR
    hipMalloc(d_cellNeighbors, numBytes * 27);                                                         CUERR

}

void gpu_h2dParticleStatics(    int nLocalParticles,
                                int nLocalCells,
                                int *h_cellId,
                                int **d_cellId,
                                int *h_id,
                                int **d_id,
                                int *h_type,
                                int **d_type,
                                realG *h_drift,
                                realG **d_drift,
                                realG *h_mass,
                                realG **d_mass,
                                bool *h_real,
                                bool **d_real,
                                int *h_cellOffsets,
                                int **d_cellOffsets,
                                int *h_particlesCell,
                                int **d_particlesCell
                                ){

    hipMemcpy(*d_cellId,  h_cellId,  nLocalParticles * sizeof(int),hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_type,  h_type,  nLocalParticles * sizeof(int),    hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_id,  h_id,  nLocalParticles * sizeof(int),    hipMemcpyHostToDevice);                              CUERR
    hipMemcpy(*d_drift, h_drift, nLocalParticles * sizeof(realG), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_mass,  h_mass,  nLocalParticles * sizeof(realG), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_real,  h_real,  nLocalParticles * sizeof(bool), hipMemcpyHostToDevice);                            CUERR
    hipMemcpy(*d_cellOffsets, h_cellOffsets, nLocalCells * sizeof(int), hipMemcpyHostToDevice);                       CUERR
    hipMemcpy(*d_particlesCell, h_particlesCell, nLocalCells * sizeof(int), hipMemcpyHostToDevice);                   CUERR
}

void gpu_h2dParticleVars(   int N,
                            realG3 *h_pos,
                            realG3 **d_pos
                        ){

    hipMemcpy(*d_pos, h_pos, sizeof(realG3) * N, hipMemcpyHostToDevice);                             CUERR

}
void gpu_d2hParticleForces( int N,
                            realG3 *h_force,
                            realG3 **d_force
                        ){
    hipDeviceSynchronize();  CUERR                        
    hipMemcpy(h_force, *d_force, sizeof(realG3) * N, hipMemcpyDeviceToHost);                         CUERR
    hipMemset(*d_force, 0, sizeof(realG3) * N);                        
}
