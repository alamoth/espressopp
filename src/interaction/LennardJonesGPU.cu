#include "hip/hip_runtime.h"
/*
  Copyright (C) 2012,2013
      Max Planck Institute for Polymer Research
  Copyright (C) 2008,2009,2010,2011
      Max-Planck-Institute for Polymer Research & Fraunhofer SCAI
  
  This file is part of ESPResSo++.
  
  ESPResSo++ is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo++ is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

// ESPP_CLASS


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "LennardJonesGPU.cuh"
#include <math.h>
#define THREADSPERBLOCK 128

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) 
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

__inline__ __device__
realG3 warpReduceSumTriple(realG3 val) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    val.x += __shfl_down_sync(0xFFFFFFFF, val.x, offset);
    val.y += __shfl_down_sync(0xFFFFFFFF, val.y, offset);
    val.z += __shfl_down_sync(0xFFFFFFFF, val.z, offset);
  }
  return val; 
}__inline__ __device__
realG warpReduceSum(realG val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down_sync(0xFFFFFFFF, val, offset);
  return val;
}
__inline__ __device__
realG blockReduceSum(realG val) {

  static __shared__ int shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSum(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0.0f;

  if (wid==0) val = warpReduceSum(val); //Final reduce within first warp

  return val;
}
__inline__ __device__
realG3 blockReduceSumTriple(realG3 val) {

  static __shared__ realG3 shared[32]; // Shared mem for 32 partial sums
  int lane = threadIdx.x % warpSize;
  int wid = threadIdx.x / warpSize;

  val = warpReduceSumTriple(val);     // Each warp performs partial reduction

  if (lane==0) shared[wid]=val; // Write reduced value to shared memory

  __syncthreads();              // Wait for all partial reductions

  //read from shared memory only if that warp existed
  val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : make_realG3(0.0, 0.0, 0.0);

  if (wid==0) val = warpReduceSumTriple(val); //Final reduce within first warp

  return val;
}

using namespace std;
#define CUERR { \
  hipError_t cudaerr; \
  if ((cudaerr = hipGetLastError()) != hipSuccess){ \
      printf("CUDA ERROR: \"%s\" in File %s at LINE %d.\n", hipGetErrorString(cudaerr), __FILE__, __LINE__); \
  } \
}

#define PRINTL { \
  if(threadIdx.x == 0){ \
    printf("Line: %d\n", __LINE__); \
  } \
}


namespace espressopp {
  namespace interaction {

    __global__ void 
    testKernel( int nPart,
                int nCells,
                int* id,
                int* cellId,
                realG4* pos,
                realG4* force,
                realG* mass,
                realG* drift,
                int* type,
                bool* real,
                int* cellParticles, 
                int* cellOffsets,
                int* cellNeighbors,
                realG* energy,
                d_LennardJonesGPU* gpuPots,
                int numPots,
                int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      
      extern __shared__ char parameter[];
      /*
      realG *s_cutoff = (realG*) &parameter[0];
      realG *s_sigma = (realG*) &parameter[sizeof(realG) * numPots];
      realG *s_epsilon = (realG*) &parameter[sizeof(realG) * 2 * numPots];
      realG *s_ff1 = (realG*) &parameter[sizeof(realG) * 3 * numPots];
      realG *s_ff2 = (realG*) &parameter[sizeof(realG) * 4 * numPots];
      
      __shared__ realG[10] cutoff;
      __shared__ realG[10] sigma;
      __shared__ realG[10] epsilon;
      __shared__ realG[10] ff1;
      __shared__ realG[10] ff2;
      __shared__ int calcMode;
      
      if(threadIdx.x == 0){
        for(int i = 0; i < numPots; ++i){
          s_cutoff[i] = gpuPots[i].cutoff;
          s_sigma[i] = gpuPots[i].sigma;
          s_epsilon[i] = gpuPots[i].epsilon;
          s_ff1[i] = gpuPots[i].ff1;
          s_ff2[i] = gpuPots[i].ff2;
        }
      }
      __syncthreads();
      */
      if(idx < nPart){
        realG3 p_pos;
        p_pos.x = pos[idx].x;
        p_pos.y = pos[idx].y;
        p_pos.z = pos[idx].z;
        //realG p_mass = mass[idx];
        //realG p_drift = drift[idx];
        int p_type = type[idx];
        //int p_real = real[idx] ? 1 : 0;
        int p_cellId = cellId[idx];
        realG3 p_force = make_realG3(0.0,0.0,0.0);
        realG3 p_dist;
        realG distSqr = 0;
        realG p_energy = 0;
        if(real[idx]){
          //#pragma unroll
          for(int i = 0; i < 27; ++i){
            int currentCellId = cellNeighbors[p_cellId * 27 + i];
            for(int j = 0; j < cellParticles[currentCellId]; ++j){
              int currentCellOffset = cellOffsets[currentCellId];
              if(currentCellOffset + j != idx){
                int potI = p_type * numPots + type[currentCellOffset + j];
                // p_dist.x = __dsub_rn(p_pos.x, pos[currentCellOffset + j].x);
                // p_dist.y = __dsub_rn(p_pos.y, pos[currentCellOffset + j].y);
                // p_dist.z = __dsub_rn(p_pos.z, pos[currentCellOffset + j].z);
                p_dist.x = p_pos.x - pos[currentCellOffset + j].x;
                p_dist.y = p_pos.y - pos[currentCellOffset + j].y;
                p_dist.z = p_pos.z - pos[currentCellOffset + j].z;
                distSqr =  p_dist.x * p_dist.x;
                distSqr += p_dist.y * p_dist.y;
                distSqr += p_dist.z * p_dist.z;
                // distSqr = 0;
                // distSqr = __fma_rn(p_dist.x, p_dist.x, distSqr);
                // distSqr = __fma_rn(p_dist.y, p_dist.y, distSqr);
                // distSqr = __fma_rn(p_dist.z, p_dist.z, distSqr);
                //if(distSqr <= (s_cutoff[potI] * s_cutoff[potI])){
                  if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
                    // if(distSqr <= __dmul_rn(gpuPots[potI].cutoff, gpuPots[potI].cutoff)){
                    if(mode == 0){
                      realG frac2 = 1.0 / distSqr;
                      // realG frac2 = __drcp_rn(distSqr);
                      realG frac6 = frac2 * frac2 * frac2;
                      // realG frac6 = __dmul_rn(frac2, __dmul_rn(frac2, frac2));
                      //realG ffactor = frac6 * (s_ff1[potI] * frac6 - s_ff2[potI]) * frac2;
                      realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                      // realG ffactor = __dmul_rn(frac6, __dmul_rn((__dsub_rn(__dmul_rn(gpuPots[potI].ff1, frac6), gpuPots[potI].ff2)), frac2));
                      // p_force.x = __fma_rn(p_dist.x, ffactor, p_force.x);
                      // p_force.y = __fma_rn(p_dist.y, ffactor, p_force.y);
                      // p_force.z = __fma_rn(p_dist.z, ffactor, p_force.z);
                      p_force.x += p_dist.x * ffactor;
                      p_force.y += p_dist.y * ffactor;
                      p_force.z += p_dist.z * ffactor;
                    }
                    if(mode == 1){
                      //realG frac2 = s_sigma[potI] * s_sigma[potI] / distSqr;
                      realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                      realG frac6 = frac2 * frac2 * frac2;
                      //realG energy = 4.0 * s_epsilon[potI] * (frac6 * frac6 - frac6);
                      realG f_energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                      p_energy += f_energy;
                    }
                }
              }
            }
          }
        }
        if(mode == 0){
          force[idx].x = real[idx] * p_force.x;
          force[idx].y = real[idx] * p_force.y;
          force[idx].z = real[idx] * p_force.z;
        }
  
        if(mode == 1){
          energy[idx] = p_energy;
        }
      }
    }
    __global__ void 
    testKernel2( const int nPart,
                const int nCells,
                const int* id,
                const int* cellId,
                const realG4* pos,
                realG4* force,
                const realG* mass,
                const realG* drift,
                const int* type,
                const bool* real,
                const int* cellParticles, 
                const int* cellOffsets,
                const int* cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* gpuPots,
                const int numPots,
                const int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      // __shared__ realG4 s_pos[THREADSPERBLOCK];
      __shared__ realG s_pos_x[THREADSPERBLOCK];
      __shared__ realG s_pos_y[THREADSPERBLOCK];
      __shared__ realG s_pos_z[THREADSPERBLOCK];
      __shared__ int s_id[THREADSPERBLOCK];
      //__shared__ realG[THREADSPERBLOCK] s_mass;
      //__shared__ realG[THREADSPERBLOCK] s_drift;
      __shared__ int s_type[THREADSPERBLOCK];
      __shared__ int activeThreads;
      // __shared__ realG s_force_x[THREADSPERBLOCK];
      // __shared__ realG s_force_y[THREADSPERBLOCK];
      // __shared__ realG s_force_z[THREADSPERBLOCK];
      __shared__ realG s_energy[THREADSPERBLOCK];
      __shared__ int numberRuns;
      int potI;
      bool sameId;
      int calcCellOffset = cellOffsets[blockIdx.x];
      // s_force_x[threadIdx.x] = 0.0f;
      // s_force_y[threadIdx.x] = 0.0f;
      // s_force_z[threadIdx.x] = 0.0f;
      realG p_energy;

      s_energy[threadIdx.x] = 0.0f;
      p_energy = 0.0f;

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }
      if(threadIdx.x == 0){
        int numberNeighborParticles = 0;
        for(int i = 0; i < 27; ++i){
            numberNeighborParticles += cellParticles[cellNeighbors[blockIdx.x * 27 + i]];
        }
        numberRuns = (numberNeighborParticles - 1) / THREADSPERBLOCK + 1;
      }

      int currentii = 0;
      int currentjj = 0;

      __syncthreads(); 

      realG3 p_dist;
      realG3 p_force;
      realG distSqr;
      for(int i = 0; i < numberRuns; ++i){
        activeThreads = 0;
        int ii,jj;
        if(threadIdx.x == 0){
          for(ii = currentii; ii < 27; ++ii){
            // if(blockIdx.x == 41 && mode == 0) {
            //   printf("accessing neighbor cell: %d\n", cellNeighbors[blockIdx.x * 27 + ii]);
            // }
            for(jj = currentjj; jj < cellParticles[cellNeighbors[blockIdx.x * 27 + ii]]; ++jj){
              if(activeThreads == THREADSPERBLOCK){ //} || (ii == 26 && jj == cellParticles[cellNeighbors[blockIdx.x * 27 + 26]] - 1)){
                //sharedMemfull = true;
                currentii = ii;
                currentjj = jj;
                goto end;
                // if(jj == cellParticles[cellNeighbors[blockIdx.x * 27 + ii]] - 1){
                //   currentii = ii+1;
                //   currentjj = 0;
                // } else {
                //   currentjj = jj + 1;
                //   currentii = ii;
                // }
              } else{
                // if(blockIdx.x == 41 && mode == 0) {
                //   printf("activeThreads %d, ii %d, jj %d, blockIdx %d\n", activeThreads, ii, jj, blockIdx.x);
                // }
                s_pos_x[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].x;
                s_pos_y[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].y;
                s_pos_z[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].z;
                s_type[activeThreads] = type[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                s_id[activeThreads] = id[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                activeThreads++;
              }
            }
            currentjj = 0;
          }
          end:;
        }
        __syncthreads();
        for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
          // s_force_x[threadIdx.x] = 0.0f;
          // s_force_y[threadIdx.x] = 0.0f;
          // s_force_z[threadIdx.x] = 0.0f;
          s_energy[threadIdx.x] = 0.0f;
          p_energy = 0.0f;
          p_force.x = 0.0f;
          p_force.y = 0.0f;
          p_force.z = 0.0f;
          potI = s_type[threadIdx.x] * numPots + type[calcCellOffset + j];
          sameId = s_id[threadIdx.x] == id[calcCellOffset + j] ? true : false;
          if(threadIdx.x < activeThreads){
            //printf("threadIdx.x=%d, idx.x: %d, own particle id: %d\n", threadIdx.x, idx, s_id[threadIdx.x]);
            // p_dist.x = s_pos[threadIdx.x].x - pos[calcCellOffset + j].x;
            // p_dist.y = s_pos[threadIdx.x].y - pos[calcCellOffset + j].y;
            // p_dist.z = s_pos[threadIdx.x].z - pos[calcCellOffset + j].z;
            p_dist.x = pos[calcCellOffset + j].x - s_pos_x[threadIdx.x];
            p_dist.y = pos[calcCellOffset + j].y - s_pos_y[threadIdx.x];
            p_dist.z = pos[calcCellOffset + j].z - s_pos_z[threadIdx.x];
            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  // s_force_x[threadIdx.x] = p_dist.x * ffactor;
                  // s_force_y[threadIdx.x] = p_dist.y * ffactor;
                  // s_force_z[threadIdx.x] = p_dist.z * ffactor;
                  p_force.x = p_dist.x * ffactor;
                  p_force.y = p_dist.y * ffactor;
                  p_force.z = p_dist.z * ffactor;
                  //printf("2. id1=%d, id2=%d %f %f %f\n", id[calcCellOffset + j], s_id[threadIdx.x], p_dist.x * ffactor,  p_dist.y * ffactor, p_dist.z * ffactor);

                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                  p_energy = energy;
                  //printf("Energy threadIdx: %d, %f\n",threadIdx.x, p_energy);
                  s_energy[threadIdx.x] = energy;
                }
              }
            }
          }
          __syncthreads();
          if(mode == 0){
            p_force = blockReduceSumTriple(p_force);
            // if(threadIdx.x == 0){
            //   for(int k = 0; k < activeThreads; ++k){
            //     force[calcCellOffset + j].x += s_force_x[k];
            //     force[calcCellOffset + j].y += s_force_y[k];
            //     force[calcCellOffset + j].z += s_force_z[k];
            //   }
            // }
            // if(threadIdx.x < activeThreads){
            //   force[calcCellOffset + j].x += s_force_x[threadIdx.x];
            //   force[calcCellOffset + j].y += s_force_y[threadIdx.x];
            //   force[calcCellOffset + j].z += s_force_z[threadIdx.x];
            if(threadIdx.x == 0){
              force[calcCellOffset + j].x += p_force.x;
              force[calcCellOffset + j].y += p_force.y;
              force[calcCellOffset + j].z += p_force.z;
            }
          }
          if(mode == 1){
            p_energy = blockReduceSum(p_energy);
            if(threadIdx.x == 0){
              // energy[calcCellOffset + j] += p_energy;
              for(int k = 0; k < activeThreads; ++k){
                energy[calcCellOffset + j] += s_energy[k];
              }
              //printf("pos in memory:: %d, Energy blockIdx: %d, Penergy: %f\n", calcCellOffset + j, blockIdx.x, energy[calcCellOffset + j]);
            }
          }
          __syncthreads();
        }
        __syncthreads();
      }
    }

    __global__ void 
    testKernel3( const int nPart,
                const int nCells,
                const int* id,
                const int* cellId,
                const realG4* pos,
                realG4* force,
                const realG* mass,
                const realG* drift,
                const int* type,
                const bool* real,
                const int* cellParticles, 
                const int* cellOffsets,
                const int* cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* gpuPots,
                const int numPots,
                const int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      __shared__ int numberLineParticles[9];
      __shared__ int numberLineWarps[9];

      int potI;
      bool sameId;
      int calcCellOffset = cellOffsets[blockIdx.x];
      

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }

      __syncthreads(); 

      realG3 p_dist;
      realG3 p_force;
      realG distSqr;
      realG t_pos_x;
      realG t_pos_y;
      realG t_pos_z;      
      realG t_type;
      realG t_id;
      realG p_energy = 0;

      int warpId = threadIdx.x / warpSize;
      int laneId = threadIdx.x % warpSize;
      if(warpId >= 9){
        printf("error");
      }
      int dataOffset = cellNeighbors[blockIdx.x * 27 + 3 * warpId];

      if(threadIdx.x < 9){
        numberLineParticles[threadIdx.x] = cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x)] + 0] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x)] + 1] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x)] + 2];
        numberLineWarps[threadIdx.x] = (numberLineParticles[threadIdx.x ] - 1) / warpSize + 1;
      }
      __syncthreads();

      for(int i = 0; i < numberLineWarps[warpId]; ++i){
        t_pos_x = 0.0f;
        t_pos_y = 0.0f;
        t_pos_z = 0.0f;
        if(i * warpSize + laneId < numberLineParticles[warpId]){
          t_pos_x = pos[dataOffset + i * warpSize + laneId].x;
          t_pos_y = pos[dataOffset + i * warpSize + laneId].y;
          t_pos_z = pos[dataOffset + i * warpSize + laneId].z;
          t_type = type[dataOffset + i * warpSize + laneId];
          t_id = id[dataOffset + i * warpSize + laneId];
      
          for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
            p_energy = 0.0f;
            p_force.x = 0.0f;
            p_force.y = 0.0f;
            p_force.z = 0.0f;
            potI = t_id * numPots + type[calcCellOffset + j];
            sameId = t_id == id[calcCellOffset + j] ? true : false;

            p_dist.x = pos[calcCellOffset + j].x - t_pos_x;
            p_dist.y = pos[calcCellOffset + j].y - t_pos_y;
            p_dist.z = pos[calcCellOffset + j].z - t_pos_z;
            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  p_force.x = p_dist.x * ffactor;
                  p_force.y = p_dist.y * ffactor;
                  p_force.z = p_dist.z * ffactor;
                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                  p_energy = energy;
                }
              }
            }
            
            __syncwarp();
            if(mode == 0){
              p_force = blockReduceSumTriple(p_force);
              if(threadIdx.x == 0){
                atomicAdd(&force[calcCellOffset + j].x, p_force.x);
                atomicAdd(&force[calcCellOffset + j].y, p_force.y);
                atomicAdd(&force[calcCellOffset + j].z, p_force.z);
                // force[calcCellOffset + j].x += p_force.x;
                // force[calcCellOffset + j].y += p_force.y;
                // force[calcCellOffset + j].z += p_force.z;
              }
            }
            if(mode == 1){
              p_energy = blockReduceSum(p_energy);
              if(threadIdx.x == 0){
                atomicAdd(&energy[calcCellOffset + j], p_energy);
                // energy[calcCellOffset + j] += p_energy;
              }
            }
          }
        }
      }
    }
    
  __global__ void
  tKern(int N, realG4* force){
    float totalForce = 0;
    for(int i=0; i<N; ++i){
      totalForce += fabs(force[i].x) + fabs(force[i].y) + fabs(force[i].z); 
      // if(force[i].x != 0.0f){
      //   printf("Force[%4d] xyz: %.8f      %.8f      %.8f\n", i, force[i].x, force[i].y, force[i].z);
      // }
    }
    printf("%f \t TotalForce\n", totalForce);
  }

  realG LJGPUdriver(StorageGPU* gpuStorage, d_LennardJonesGPU* gpuPots, int mode){
    int numThreads = 128;
    int numBlocks = (gpuStorage->numberLocalParticles) / numThreads + 1;
    realG *h_energy; 
    realG *d_energy;
    realG totalEnergy = 0;

    h_energy = new realG[gpuStorage->numberLocalParticles];
    hipMalloc(&d_energy, sizeof(realG) * gpuStorage->numberLocalParticles);
    hipMemset(d_energy, 0, sizeof(realG) * gpuStorage->numberLocalParticles);
    unsigned numPots = 1;
    unsigned shared_mem_size = 10 * sizeof(realG) * 5;
    hipMemset(gpuStorage->d_force, 0, sizeof(realG4) * gpuStorage->numberLocalParticles);
    if(false){
      testKernel<<<numBlocks, numThreads, shared_mem_size>>>(
                              gpuStorage->numberLocalParticles, 
                              gpuStorage->numberLocalCells, 
                              gpuStorage->d_id,
                              gpuStorage->d_cellId,
                              gpuStorage->d_pos,
                              gpuStorage->d_force,
                              gpuStorage->d_mass,
                              gpuStorage->d_drift,
                              gpuStorage->d_type,
                              gpuStorage->d_real,
                              gpuStorage->d_particlesCell,
                              gpuStorage->d_cellOffsets,
                              gpuStorage->d_cellNeighbors,
                              d_energy,
                              gpuPots,
                              numPots,
                              mode
                            );
    } else{
      testKernel3<<<gpuStorage->numberLocalCells, THREADSPERBLOCK>>>(
        gpuStorage->numberLocalParticles, 
        gpuStorage->numberLocalCells, 
        gpuStorage->d_id,
        gpuStorage->d_cellId,
        gpuStorage->d_pos,
        gpuStorage->d_force,
        gpuStorage->d_mass,
        gpuStorage->d_drift,
        gpuStorage->d_type,
        gpuStorage->d_real,
        gpuStorage->d_particlesCell,
        gpuStorage->d_cellOffsets,
        gpuStorage->d_cellNeighbors,
        d_energy,
        gpuPots,
        numPots,
        mode
      ); CUERR
    }
    //hipDeviceSynchronize();

      //printf("---\n");
      if(mode == 1) {
        hipMemcpy(h_energy, d_energy, sizeof(realG) * gpuStorage->numberLocalParticles, hipMemcpyDeviceToHost); CUERR
        for (int i = 0; i < gpuStorage->numberLocalParticles; ++i){ 
          totalEnergy += h_energy[i];
        }
      }

      return totalEnergy / (double)2.0f;
    }
  }
}

