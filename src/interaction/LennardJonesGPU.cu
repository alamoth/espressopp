/*
  Copyright (C) 2012,2013
      Max Planck Institute for Polymer Research
  Copyright (C) 2008,2009,2010,2011
      Max-Planck-Institute for Polymer Research & Fraunhofer SCAI
  
  This file is part of ESPResSo++.
  
  ESPResSo++ is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo++ is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

// ESPP_CLASS


#include <hip/hip_runtime.h>
#include "LennardJonesGPU.hpp"
#include <stdio.h>


using namespace std;


namespace espressopp {
  namespace interaction {

    void d_LennardJonesGPU::testFF(d_LennardJonesGPU* potential){
      printf("bna %f\n", potential->getSigma());
    }
  }
}


