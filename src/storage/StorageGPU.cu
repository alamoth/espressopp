#include "hip/hip_runtime.h"
#include "StorageGPU.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUERR { \
    hipError_t cudaerr; \
    if ((cudaerr = hipGetLastError()) != hipSuccess){ \
        printf("CUDA ERROR: \"%s\" in File %s at LINE %d.\n", hipGetErrorString(cudaerr), __FILE__, __LINE__); \
    } \
}

void gpu_resizeParticleData(    int N, 
                                int **d_cellId,
                                int **d_id,
                                int **d_type, 
                                double **d_drift,
                                double **d_mass, 
                                double3 **d_pos,
                                double3 **d_force,
                                bool **d_real
                            ){
    int numBytesD = N * sizeof(double);
    int numBytesI = N * sizeof(int);

    if(*d_cellId != 0) hipFree(*d_cellId);                                                             CUERR
    if(*d_id != 0) hipFree(*d_id);                                                                     CUERR
    if(*d_type != 0) hipFree(*d_type);                                                                 CUERR
    if(*d_drift != 0) hipFree(*d_drift);                                                               CUERR
    if(*d_mass != 0) hipFree(*d_mass);                                                                 CUERR
    if(*d_pos != 0) hipFree(*d_pos);                                                                   CUERR
    if(*d_force != 0) hipFree(*d_force);                                                               CUERR
    if(*d_real != 0) hipFree(*d_real);                                                                 CUERR

    hipMalloc(d_cellId, numBytesI);                                                                    CUERR 
    hipMalloc(d_id, numBytesI);                                                                        CUERR 
    hipMalloc(d_type, numBytesI);                                                                      CUERR
    hipMalloc(d_pos,   sizeof(double3) * N);                                                           CUERR
    hipMalloc(d_force, sizeof(double3) * N);                                                           CUERR
    hipMalloc(d_mass, numBytesD);                                                                      CUERR
    hipMalloc(d_drift, numBytesD);                                                                     CUERR
    hipMalloc(d_real, sizeof(bool) * N);                                                               CUERR

    hipMemset(*d_force, 0, sizeof(double3) * N);                                                       CUERR
    
}

void gpu_h2dCellData(   int M, 
                        int *h_cellNeighbors,
                        int **d_cellNeighbors
                    ) {
    int numBytesI = M * sizeof(int);
    hipMemcpy(*d_cellNeighbors, h_cellNeighbors, numBytesI * 27, hipMemcpyHostToDevice);              CUERR
}

void gpu_resizeCellData(    int M,
                            int **d_cellOffsets,
                            int **d_particlesCell,
                            int **d_cellNeighbors) {

    int numBytes = M * sizeof(int);

    if(*d_cellOffsets != 0 && *d_particlesCell != 0){
        hipFree(*d_cellOffsets);                                                                       CUERR
        hipFree(*d_particlesCell);                                                                     CUERR
        hipFree(*d_cellNeighbors);                                                                     CUERR
    }

    hipMalloc(d_cellOffsets, numBytes);                                                                CUERR
    hipMalloc(d_particlesCell, numBytes);                                                              CUERR
    hipMalloc(d_cellNeighbors, numBytes * 27);                                                         CUERR

}

void gpu_h2dParticleStatics(    int nLocalParticles,
                                int nLocalCells,
                                int *h_cellId,
                                int **d_cellId,
                                int *h_id,
                                int **d_id,
                                int *h_type,
                                int **d_type,
                                double *h_drift,
                                double **d_drift,
                                double *h_mass,
                                double **d_mass,
                                bool *h_real,
                                bool **d_real,
                                int *h_cellOffsets,
                                int **d_cellOffsets,
                                int *h_particlesCell,
                                int **d_particlesCell
                                ){

    hipMemcpy(*d_cellId,  h_cellId,  nLocalParticles * sizeof(int),hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_type,  h_type,  nLocalParticles * sizeof(int),    hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_id,  h_id,  nLocalParticles * sizeof(int),    hipMemcpyHostToDevice);                              CUERR
    hipMemcpy(*d_drift, h_drift, nLocalParticles * sizeof(double), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_mass,  h_mass,  nLocalParticles * sizeof(double), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_real,  h_real,  nLocalParticles * sizeof(bool), hipMemcpyHostToDevice);                            CUERR
    hipMemcpy(*d_cellOffsets, h_cellOffsets, nLocalCells * sizeof(int), hipMemcpyHostToDevice);                       CUERR
    hipMemcpy(*d_particlesCell, h_particlesCell, nLocalCells * sizeof(int), hipMemcpyHostToDevice);                   CUERR
}

void gpu_h2dParticleVars(   int N,
                            double3 *h_pos,
                            double3 **d_pos
                        ){

    hipMemcpy(*d_pos, h_pos, sizeof(double3) * N, hipMemcpyHostToDevice);                             CUERR

}
void gpu_d2hParticleForces( int N,
                            double3 *h_force,
                            double3 **d_force
                        ){
    hipMemcpy(h_force, *d_force, sizeof(double3) * N, hipMemcpyDeviceToHost);                         CUERR
}
