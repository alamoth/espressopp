#include "hip/hip_runtime.h"
#include "StorageGPU.cuh"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUERR { \
    hipError_t cudaerr; \
    if ((cudaerr = hipGetLastError()) != hipSuccess){ \
        printf("CUDA ERROR: \"%s\" in File %s at LINE %d.\n", hipGetErrorString(cudaerr), __FILE__, __LINE__); \
    } \
}

void gpu_resizeParticleData(    int N, 
                                int **d_cellId,
                                int **d_id,
                                int **d_type, 
                                double **d_drift,
                                double **d_mass, 
                                double3 **d_pos,
                                double3 **d_force,
                                bool **d_ghost
                            ){
    int numBytesD = N * sizeof(double);
    int numBytesI = N * sizeof(int);

    if(*d_cellId != 0) hipFree(*d_cellId);                                                             CUERR
    if(*d_id != 0) hipFree(*d_id);                                                             CUERR
    if(*d_type != 0) hipFree(*d_type);                                                                 CUERR
    if(*d_drift != 0) hipFree(*d_drift);                                                               CUERR
    if(*d_mass != 0) hipFree(*d_mass);                                                                 CUERR
    if(*d_pos != 0) hipFree(*d_pos);                                                                   CUERR
    if(*d_force != 0) hipFree(*d_force);                                                               CUERR
    if(*d_ghost != 0) hipFree(*d_ghost);                                                               CUERR

    hipMalloc(d_cellId, numBytesI);                                                                    CUERR 
    hipMalloc(d_id, numBytesI);                                                                    CUERR 
    hipMalloc(d_type, numBytesI);                                                                      CUERR
    hipMalloc(d_pos,   sizeof(double3) * N);                                                           CUERR
    hipMalloc(d_force, sizeof(double3) * N);                                                           CUERR
    hipMalloc(d_mass, numBytesD);                                                                      CUERR
    hipMalloc(d_drift, numBytesD);                                                                     CUERR
    hipMalloc(d_ghost, sizeof(bool) * N);                                                              CUERR

    hipMemset(*d_force, 0, sizeof(double3) * N);                                                       CUERR
    
}

void gpu_h2dCellData(   int M, 
                        int **d_cellOffsets, 
                        int **d_numberCellNeighbors, 
                        int *h_cellOffsets, 
                        int *h_numberCellNeighbors) {
    int numBytesI = M * sizeof(int);
    hipMemcpy(*d_cellOffsets, h_cellOffsets, numBytesI, hipMemcpyHostToDevice);                       CUERR
    hipMemcpy(*d_numberCellNeighbors, h_numberCellNeighbors, numBytesI, hipMemcpyHostToDevice);       CUERR

}

void gpu_resizeCellData(    int M,
                            int **d_cellOffsets,
                            int **d_numberCellNeighbors) {

    int numBytes = M * sizeof(int);

    if(*d_cellOffsets != 0 && *d_numberCellNeighbors != 0){
        hipFree(*d_cellOffsets);                                                                       CUERR
        hipFree(*d_numberCellNeighbors);                                                               CUERR
    }

    hipMalloc(d_cellOffsets, numBytes);                                                                CUERR
    hipMalloc(d_numberCellNeighbors, numBytes);                                                        CUERR

}

void gpu_h2dParticleStatics(    int N,
                                int *h_cellId,
                                int **d_cellId,
                                int *h_id,
                                int **d_id,
                                int *h_type,
                                int **d_type,
                                double *h_drift,
                                double **d_drift,
                                double *h_mass,
                                double **d_mass,
                                bool *h_ghost,
                                bool **d_ghost){

    hipMemcpy(*d_cellId,  h_cellId,  N * sizeof(int),hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_type,  h_type,  N * sizeof(int),    hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_id,  h_id,  N * sizeof(int),    hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_drift, h_drift, N * sizeof(double), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_mass,  h_mass,  N * sizeof(double), hipMemcpyHostToDevice);                          CUERR
    hipMemcpy(*d_ghost,  h_ghost,  N * sizeof(bool), hipMemcpyHostToDevice);                          CUERR
}

void gpu_h2dParticleVars(   int N,
                            double3 *h_pos,
                            double3 **d_pos
                        ){

    hipMemcpy(*d_pos, h_pos, sizeof(double3) * N, hipMemcpyHostToDevice);                             CUERR

}
void gpu_d2hParticleForces( int N,
                            double3 *h_force,
                            double3 **d_force
                        ){
    hipMemcpy(h_force, *d_force, sizeof(double3) * N, hipMemcpyDeviceToHost);                         CUERR
}
