#include "hip/hip_runtime.h"
/*
  Copyright (C) 2012,2013
      Max Planck Institute for Polymer Research
  Copyright (C) 2008,2009,2010,2011
      Max-Planck-Institute for Polymer Research & Fraunhofer SCAI
  
  This file is part of ESPResSo++.
  
  ESPResSo++ is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo++ is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

// ESPP_CLASS


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "LennardJonesGPU.cuh"
#include <math.h>
#include <assert.h>
#define THREADSPERBLOCK 1024
//#ifdef __NVCC__

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) 
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

template<typename T>
__device__ __forceinline__ T ldg(const T* ptr) {
return *ptr;
}


namespace espressopp {
  namespace interaction {

// Kernel using Verlet list
__global__ void 
    verletListKernel( const int nPart,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                realG* energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode,
                const int* __restrict__ vl,
                const int* __restrict__ num_nb){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      extern __shared__ char parameter[];
      
      realG *s_cutoff = (realG*) &parameter[0];
      realG *s_sigma = (realG*) &parameter[sizeof(realG) * numPots];
      realG *s_epsilon = (realG*) &parameter[sizeof(realG) * 2 * numPots];
      realG *s_ff1 = (realG*) &parameter[sizeof(realG) * 3 * numPots];
      realG *s_ff2 = (realG*) &parameter[sizeof(realG) * 4 * numPots];
            
      if(threadIdx.x < numPots){
          s_cutoff[threadIdx.x] = gpuPots[threadIdx.x].cutoff;
          s_sigma[threadIdx.x] = gpuPots[threadIdx.x].sigma;
          s_epsilon[threadIdx.x] = gpuPots[threadIdx.x].epsilon;
          s_ff1[threadIdx.x] = gpuPots[threadIdx.x].ff1;
          s_ff2[threadIdx.x] = gpuPots[threadIdx.x].ff2;
      }
      __syncthreads();
      
      
      if (idx >= nPart) return;
      if (!real[idx]) return;

      realG3 p_pos = pos[idx];
      int p_type = type[idx];
      //realG p_mass = mass[idx];
      //realG p_drift = drift[idx];
      realG3 p_force = make_realG3(0.0,0.0,0.0);
      realG p_energy = 0;
      int n_nb = num_nb[idx];

      // Iterate particle specific Verlet list
      for(int i = 0; i < n_nb; ++i){
        // Load particle data into thread memory and calculate squared distance 
        int p2_idx = vl[i * nPart + idx];
        realG3 p_dist;
        realG3 p2_pos = pos[p2_idx];
        int potIdx = p_type * numPots + type[p2_idx];
        p_dist = p_pos - p2_pos;
        realG distSqr = dot(p_dist, p_dist);
        if(distSqr <= (s_cutoff[potIdx] * s_cutoff[potIdx])){
          if(mode == 0){
            // Inline force calculation
            realG frac2 = 1.0 / distSqr;
            realG frac6 = frac2 * frac2 * frac2;
            realG calcResult = frac6 * (s_ff1[potIdx] * frac6 - s_ff2[potIdx]) * frac2;
            p_force += p_dist * calcResult;
          }
          if(mode == 1){
            // Inline energy calculation
            realG frac2 = s_sigma[potIdx] * s_sigma[potIdx] / distSqr;
            realG frac6 = frac2 * frac2 * frac2;
            realG calcResult = 4.0 * s_epsilon[potIdx] * (frac6 * frac6 - frac6);
            p_energy += calcResult;
          }
        }
      }

      if(mode == 0){
        force[idx] = p_force;
      }

      if(mode == 1){
        energy[idx] = p_energy;
      }
    }

    // Kernel not using Verlet list, OTP
    __global__ void 
    testKernel( const int nPart,
                const int nCells,
                const int* __restrict__ id,
                const int* __restrict__ cellId,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                const int* __restrict__ cellParticlesN, 
                const int* __restrict__ cellOffsets,
                const int* __restrict__ cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      extern __shared__ char parameter[];
      
      realG *s_cutoff = (realG*) &parameter[0];
      realG *s_sigma = (realG*) &parameter[sizeof(realG) * numPots];
      realG *s_epsilon = (realG*) &parameter[sizeof(realG) * 2 * numPots];
      realG *s_ff1 = (realG*) &parameter[sizeof(realG) * 3 * numPots];
      realG *s_ff2 = (realG*) &parameter[sizeof(realG) * 4 * numPots];
            
      if(threadIdx.x < numPots){
          s_cutoff[threadIdx.x] = gpuPots[threadIdx.x].cutoff;
          s_sigma[threadIdx.x] = gpuPots[threadIdx.x].sigma;
          s_epsilon[threadIdx.x] = gpuPots[threadIdx.x].epsilon;
          s_ff1[threadIdx.x] = gpuPots[threadIdx.x].ff1;
          s_ff2[threadIdx.x] = gpuPots[threadIdx.x].ff2;
      }

      if (idx >= nPart) return;
      if (!real[idx]) return;
      __syncthreads();

      realG3 p_pos = pos[idx];
      //realG p_mass = mass[idx];
      //realG p_drift = drift[idx];
      int p_type = type[idx];
      int p_cellId = cellId[idx];
      realG3 p_force = make_realG3(0.0,0.0,0.0);
      realG p_energy = 0;
      // Iterate over all 27 required cells
      for(int i = 0; i < 27; ++i){
        // Load cell metadata
        int currentCellId = *(cellNeighbors+(p_cellId * 27 + i));
        int sizeCell = *(cellParticlesN+currentCellId);
        int currentCellOffset = *(cellOffsets+currentCellId);
        // Iterate over all particles inside the cell
        for(int j = 0; j < sizeCell; ++j){
          int pOffset = currentCellOffset + j;
          if(pOffset != idx){ // If particles are not the same
            // Load second particle data
            int potIdx = p_type * numPots + *(type+pOffset);
            realG3 p2_pos = pos[pOffset];
            realG3 distVec = p_pos - p2_pos;
            realG distSqr = dot(distVec, distVec);
            if(distSqr <= (s_cutoff[potIdx] * s_cutoff[potIdx])){
              if(mode == 0){
                realG frac2 = 1.0 / distSqr;
                realG frac6 = frac2 * frac2 * frac2;
                realG calcResult = frac6 * (s_ff1[potIdx] * frac6 - s_ff2[potIdx]) * frac2;
                p_force += distVec * calcResult;
              }
              if(mode == 1){
                realG frac2 = s_sigma[potIdx] * s_sigma[potIdx] / distSqr;
                realG frac6 = frac2 * frac2 * frac2;
                realG calcResult = 4.0 * s_epsilon[potIdx] * (frac6 * frac6 - frac6);
                p_energy += calcResult;
              }
            }
          }
          // __syncwarp();
        }
        // __syncwarp();
      }
      if(mode == 0){
        force[idx] = p_force;
      }

      if(mode == 1){
        energy[idx] = p_energy;
      }
    }

    // OBC kernel version 1
    __global__ void 
    testKernel2( const int nPart,
                const int nCells,
                const int* id,
                const int* cellId,
                const realG3* pos,
                realG3* force,
                const realG* mass,
                const realG* drift,
                const int* type,
                const bool* real,
                const int* cellParticles, 
                const int* cellOffsets,
                const int* cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* gpuPots,
                const int numPots,
                const int mode){
      __shared__ realG s_pos_x[THREADSPERBLOCK];
      __shared__ realG s_pos_y[THREADSPERBLOCK];
      __shared__ realG s_pos_z[THREADSPERBLOCK];
      __shared__ int s_id[THREADSPERBLOCK];
      //__shared__ realG[THREADSPERBLOCK] s_mass;
      //__shared__ realG[THREADSPERBLOCK] s_drift;
      __shared__ int s_type[THREADSPERBLOCK];
      __shared__ int activeThreads;
      __shared__ realG s_energy[THREADSPERBLOCK];
      __shared__ int numberRuns;
      int potI;
      bool sameId;
      int calcCellOffset = cellOffsets[blockIdx.x];
      realG p_energy;

      s_energy[threadIdx.x] = 0.0f;
      p_energy = 0.0f;

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }

      // Thread 0 sums up all neighbor particles
      if(threadIdx.x == 0){
        int numberNeighborParticles = 0;
        for(int i = 0; i < 27; ++i){
            numberNeighborParticles += cellParticles[cellNeighbors[blockIdx.x * 27 + i]];
        }
        numberRuns = (numberNeighborParticles - 1) / THREADSPERBLOCK + 1;
      }

      int currentii = 0;
      int currentjj = 0;

      __syncthreads(); 

      realG3 p_dist;
      realG3 p_force;
      realG distSqr;
      for(int i = 0; i < numberRuns; ++i){
        activeThreads = 0;
        int ii,jj;
        // Workaround with goto to address and load neighbor particles into shared memory
        if(threadIdx.x == 0){
          for(ii = currentii; ii < 27; ++ii){
            for(jj = currentjj; jj < cellParticles[cellNeighbors[blockIdx.x * 27 + ii]]; ++jj){
              if(activeThreads == THREADSPERBLOCK){
                currentii = ii;
                currentjj = jj;
                goto end;
              } else{
                s_pos_x[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].x;
                s_pos_y[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].y;
                s_pos_z[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].z;
                s_type[activeThreads] = type[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                s_id[activeThreads] = id[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                activeThreads++;
              }
            }
            currentjj = 0;
          }
          end:;
        }
        __syncthreads();
        // Iterate over neighbor particles
        for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
          s_energy[threadIdx.x] = 0.0f;
          p_energy = 0.0f;
          p_force.x = 0.0f;
          p_force.y = 0.0f;
          p_force.z = 0.0f;
          potI = s_type[threadIdx.x] * numPots + type[calcCellOffset + j];
          sameId = s_id[threadIdx.x] == id[calcCellOffset + j] ? true : false;
          if(threadIdx.x < activeThreads){
            p_dist.x = pos[calcCellOffset + j].x - s_pos_x[threadIdx.x];
            p_dist.y = pos[calcCellOffset + j].y - s_pos_y[threadIdx.x];
            p_dist.z = pos[calcCellOffset + j].z - s_pos_z[threadIdx.x];
            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  p_force.x = p_dist.x * ffactor;
                  p_force.y = p_dist.y * ffactor;
                  p_force.z = p_dist.z * ffactor;

                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                  p_energy = energy;
                  s_energy[threadIdx.x] = energy;
                }
              }
            }
          }
          __syncthreads();
          if(mode == 0){
            p_force = blockReduceSumTriple(p_force, 0xffffffff);
            if(threadIdx.x == 0){
              force[calcCellOffset + j].x += p_force.x;
              force[calcCellOffset + j].y += p_force.y;
              force[calcCellOffset + j].z += p_force.z;
            }
          }
          if(mode == 1){
            p_energy = blockReduceSum(p_energy, 0xffffffff);
            if(threadIdx.x == 0){
              for(int k = 0; k < activeThreads; ++k){
                energy[calcCellOffset + j] += s_energy[k];
              }
            }
          }
          __syncthreads();
        }
        __syncthreads();
      }
    }

    // OBC version 2, warp approach
    __global__ void 
    testKernel3(const int nPart,
                const int nCells,
                const int* __restrict__ id,
                const int* __restrict__ cellId,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                const int* __restrict__ cellParticles, 
                const int* __restrict__ cellOffsets,
                const int* __restrict__ cellNeighbors,
                realG* __restrict__ energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode){
      __shared__ int numberLineParticles[9];
      __shared__ int numberLineWarps[9];

      int calcCellOffset = cellOffsets[blockIdx.x];

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }


      int warpId = threadIdx.x / warpSize;
      int laneId = threadIdx.x % warpSize;

      int dataOffset = cellOffsets[cellNeighbors[blockIdx.x * 27 + 3 * warpId]];

      // Calculate total number of particles in a lane, each assigned to one warp
      if(threadIdx.x < 9){
        numberLineParticles[threadIdx.x] = cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 0]] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 1]] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 2]];
        numberLineWarps[threadIdx.x] = (numberLineParticles[threadIdx.x ] - 1) / warpSize + 1;
      }
      __syncthreads();

      // Iterate though all particles on each line
      for(int i = 0; i < numberLineWarps[warpId]; ++i){
        // Since not all threads inside a warp are active, calculate the mask for the reduction
        unsigned mask = __ballot_sync(0xffffffff, i * warpSize + laneId < numberLineParticles[warpId]);
        if(i * warpSize + laneId < numberLineParticles[warpId]){
          realG3 t_pos = pos[dataOffset + i * warpSize + laneId];
          int t_type = type[dataOffset + i * warpSize + laneId];
          int t_id = id[dataOffset + i * warpSize + laneId];
          for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
            realG3 p_force;
            realG p_energy;
            p_energy = 0.0f;
            p_force.x = 0.0f;
            p_force.y = 0.0f;
            p_force.z = 0.0f;
            int potI = t_type * numPots + type[calcCellOffset + j];
            bool sameId = t_id == id[calcCellOffset + j] ? true : false;

            realG3 p_dist;
            p_dist = pos[calcCellOffset + j] - t_pos;

            realG distSqr =  dot(p_dist, p_dist);
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  p_force = p_dist * ffactor;
                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  p_energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                }
              }
            }
          
            // __syncthreads();
            //__syncwarp();
            if(mode == 0){
              p_force = warpReduceSumTriple(p_force, mask);
              if(laneId == 0){
                atomicAdd(&force[calcCellOffset + j].x, p_force.x);
                atomicAdd(&force[calcCellOffset + j].y, p_force.y);
                atomicAdd(&force[calcCellOffset + j].z, p_force.z);
              }
            }
            if(mode == 1){
              p_energy = warpReduceSum(p_energy, mask);
              if(laneId == 0){
                  atomicAdd(&energy[calcCellOffset + j], p_energy);
              }
            }
            __syncwarp();
          }
        }
      }
    }
    
  realG LJGPUdriverVl(StorageGPU* gpuStorage, d_LennardJonesGPU* gpuPots, int ptypes, int* vl, int* n_nb, int mode){
    realG *h_energy; 
    realG *d_energy;
    realG totalEnergy = 0;

    if(mode == 1) {
      h_energy = new realG[gpuStorage->numberLocalParticles];
      hipMalloc(&d_energy, sizeof(realG) * gpuStorage->numberLocalParticles); CUERR
      hipMemset(d_energy, 0, sizeof(realG) * gpuStorage->numberLocalParticles); CUERR
      hipMemset(gpuStorage->d_force, 0, sizeof(realG3) * gpuStorage->numberLocalParticles); CUERR
    }
    
    unsigned shared_mem_size = ptypes * ptypes * sizeof(realG) * 5;
    // hipEvent_t start, stop;
    // hipEventCreate(&start); CUERR
    // hipEventCreate(&stop); CUERR
    // hipEventRecord(start); CUERR

    verletListKernel<<<SDIV(gpuStorage->numberLocalParticles, THREADSPERBLOCK), THREADSPERBLOCK, shared_mem_size>>>(
      gpuStorage->numberLocalParticles, 
      gpuStorage->d_pos,
      gpuStorage->d_force,
      gpuStorage->d_mass,
      gpuStorage->d_drift,
      gpuStorage->d_type,
      gpuStorage->d_real,
      d_energy,
      gpuPots,
      ptypes,
      mode,
      vl,
      n_nb
    );

    // hipDeviceSynchronize(); CUERR
    // hipEventRecord(stop); CUERR
    // hipEventSynchronize(stop); CUERR
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop); CUERR
    // printf("%s kernel time: %2.6f\n", mode==0? "Force" : "Energy", milliseconds);
      if(mode == 1) {
        hipMemcpy(h_energy, d_energy, sizeof(realG) * gpuStorage->numberLocalParticles, hipMemcpyDeviceToHost); CUERR
        for (int i = 0; i < gpuStorage->numberLocalParticles; ++i){ 
          totalEnergy += h_energy[i];
        }
        hipFree(d_energy);
      }
      return totalEnergy / (double)2.0;

  }

  realG LJGPUdriver(StorageGPU* gpuStorage, d_LennardJonesGPU* gpuPots, int ptypes, int mode){
    realG totalEnergy = 0;
    realG *h_energy; 
    realG *d_energy;
    if(mode == 1) {      
      h_energy = new realG[gpuStorage->numberLocalParticles];
      hipMalloc(&d_energy, sizeof(realG) * gpuStorage->numberLocalParticles); CUERR
      hipMemset(d_energy, 0, sizeof(realG) * gpuStorage->numberLocalParticles); CUERR
    }
    // hipMemset(gpuStorage->d_force, 0, sizeof(realG3) * gpuStorage->numberLocalParticles); CUERR
    unsigned shared_mem_size = ptypes * sizeof(realG) * 5;
    // hipEvent_t start, stop;
    // hipEventCreate(&start); CUERR
    // hipEventCreate(&stop); CUERR
    // hipEventRecord(start); CUERR
    testKernel<<<SDIV(gpuStorage->numberLocalParticles, THREADSPERBLOCK), THREADSPERBLOCK, shared_mem_size>>>(
      //  testKernel2<<<gpuStorage->numberLocalCells, THREADSPERBLOCK>>>(
      // testKernel3<<<gpuStorage->numberLocalCells, 288>>>(
                            gpuStorage->numberLocalParticles, 
                            gpuStorage->numberLocalCells, 
                            gpuStorage->d_id,
                            gpuStorage->d_cellId,
                            gpuStorage->d_pos,
                            gpuStorage->d_force,
                            gpuStorage->d_mass,
                            gpuStorage->d_drift,
                            gpuStorage->d_type,
                            gpuStorage->d_real,
                            gpuStorage->d_particlesCell,
                            gpuStorage->d_cellOffsets,
                            gpuStorage->d_cellNeighbors,
                            d_energy,
                            gpuPots,
                            ptypes,
                            mode
                          );
  
    // hipDeviceSynchronize(); CUERR
    // hipEventRecord(stop); CUERR
    // hipEventSynchronize(stop); CUERR
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, start, stop); CUERR
    // printf("%s kernel time: %2.6f\n", mode==0? "Force" : "Energy", milliseconds);
    if(mode == 1) {
      hipMemcpy(h_energy, d_energy, sizeof(realG) * gpuStorage->numberLocalParticles, hipMemcpyDeviceToHost); CUERR
      for (int i = 0; i < gpuStorage->numberLocalParticles; ++i){ 
        totalEnergy += h_energy[i];
      }
      hipFree(d_energy);
    }
    return totalEnergy / (double)2.0;
    }
  }
}

// #endif