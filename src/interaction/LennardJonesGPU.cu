#include "hip/hip_runtime.h"
/*
  Copyright (C) 2012,2013
      Max Planck Institute for Polymer Research
  Copyright (C) 2008,2009,2010,2011
      Max-Planck-Institute for Polymer Research & Fraunhofer SCAI
  
  This file is part of ESPResSo++.
  
  ESPResSo++ is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.
  
  ESPResSo++ is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.
  
  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>. 
*/

// ESPP_CLASS


#include <hip/hip_runtime.h>
#include <stdio.h>
#include "LennardJonesGPU.cuh"
#include <math.h>
#include <assert.h>
#define THREADSPERBLOCK 128
//#ifdef __NVCC__

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) 
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

namespace espressopp {
  namespace interaction {

__global__ void 
    verletListKernel( const int nPart,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                realG* energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode,
                const int* __restrict__ vl,
                const int* __restrict__ num_nb){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      extern __shared__ char parameter[];
      
      realG *s_cutoff = (realG*) &parameter[0];
      realG *s_sigma = (realG*) &parameter[sizeof(realG) * numPots];
      realG *s_epsilon = (realG*) &parameter[sizeof(realG) * 2 * numPots];
      realG *s_ff1 = (realG*) &parameter[sizeof(realG) * 3 * numPots];
      realG *s_ff2 = (realG*) &parameter[sizeof(realG) * 4 * numPots];
            
      if(threadIdx.x < numPots){
          s_cutoff[threadIdx.x] = gpuPots[threadIdx.x].cutoff;
          s_sigma[threadIdx.x] = gpuPots[threadIdx.x].sigma;
          s_epsilon[threadIdx.x] = gpuPots[threadIdx.x].epsilon;
          s_ff1[threadIdx.x] = gpuPots[threadIdx.x].ff1;
          s_ff2[threadIdx.x] = gpuPots[threadIdx.x].ff2;
      }
      __syncthreads();
      
      
      if (idx >= nPart) return;
      if (!real[idx]) return;

      realG3 p_pos = pos[idx];
      realG3 p2_pos;
      int p_type = type[idx];
      //realG p_mass = mass[idx];
      //realG p_drift = drift[idx];
      // int p_cellId = cellId[idx];
      realG3 p_force = make_realG3(0.0,0.0,0.0,0.0);
      realG3 p_dist;
      realG distSqr = 0;
      realG p_energy = 0;
      realG frac2;
      realG frac6;
      realG calcResult;
      int currentCellId;
      int n_nb = num_nb[idx];
      int potIdx;
      //#pragma unroll

      for(int i = 0; i < n_nb; ++i){
        int p2_idx = vl[i * nPart + idx];
        
        assert(p2_idx != idx);
        assert(p2_idx < nPart);
        // if(p2_idx == -1){
        //   printf("idx: %d, i: %d, numMb: %d\n", idx, i, n_nb);
        // }
        assert(p2_idx != -1);

        potIdx = p_type * numPots + type[p2_idx];
        p2_pos = pos[p2_idx];

        p_dist.x = p_pos.x - p2_pos.x;
        p_dist.y = p_pos.y - p2_pos.y;
        p_dist.z = p_pos.z - p2_pos.z;
        distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
        if(distSqr <= (s_cutoff[potIdx] * s_cutoff[potIdx])){
          if(mode == 0){
            frac2 = 1.0 / distSqr;
            frac6 = frac2 * frac2 * frac2;
            calcResult = frac6 * (s_ff1[potIdx] * frac6 - s_ff2[potIdx]) * frac2;
            p_force.x += p_dist.x * calcResult;
            p_force.y += p_dist.y * calcResult;
            p_force.z += p_dist.z * calcResult;
            // printf("Particle[%d] check against %d (%s)\n", idx, p2_idx, real[p2_idx]? "real":"ghost");

          }
          if(mode == 1){
            frac2 = s_sigma[potIdx] * s_sigma[potIdx] / distSqr;
            frac6 = frac2 * frac2 * frac2;
            calcResult = 4.0 * s_epsilon[potIdx] * (frac6 * frac6 - frac6);
            p_energy += calcResult;
          }
        }
      }

      if(mode == 0){
        // p_force.x *= real[idx];
        // p_force.y *= real[idx];
        // p_force.z *= real[idx];

        force[idx] = p_force;
      }

      if(mode == 1){
        energy[idx] = p_energy;
      }
    }
    __global__ void 
    testKernel( const int nPart,
                const int nCells,
                const int* __restrict__ id,
                const int* __restrict__ cellId,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                const int* __restrict__ cellParticles, 
                const int* __restrict__ cellOffsets,
                const int* __restrict__ cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;

      extern __shared__ char parameter[];
      
      realG *s_cutoff = (realG*) &parameter[0];
      realG *s_sigma = (realG*) &parameter[sizeof(realG) * numPots];
      realG *s_epsilon = (realG*) &parameter[sizeof(realG) * 2 * numPots];
      realG *s_ff1 = (realG*) &parameter[sizeof(realG) * 3 * numPots];
      realG *s_ff2 = (realG*) &parameter[sizeof(realG) * 4 * numPots];
            
      if(threadIdx.x < numPots){
          s_cutoff[threadIdx.x] = gpuPots[threadIdx.x].cutoff;
          s_sigma[threadIdx.x] = gpuPots[threadIdx.x].sigma;
          s_epsilon[threadIdx.x] = gpuPots[threadIdx.x].epsilon;
          s_ff1[threadIdx.x] = gpuPots[threadIdx.x].ff1;
          s_ff2[threadIdx.x] = gpuPots[threadIdx.x].ff2;
      }
      __syncthreads();
      
      
      if (idx >= nPart) return;
      if (!real[idx]) return;

      realG3 p_pos = pos[idx];
      //realG p_mass = mass[idx];
      //realG p_drift = drift[idx];
      int p_type = type[idx];
      //int p_real = real[idx] ? 1 : 0;
      int p_cellId = cellId[idx];
      realG3 p_force = make_realG3(0.0,0.0,0.0,0.0);
      realG3 p_dist;
      realG distSqr = 0;
      realG p_energy = 0;
      realG frac2;
      realG frac6;
      realG calcResult;
      int currentCellId;
      //#pragma unroll
      for(int i = 0; i < 27; ++i){
        currentCellId = cellNeighbors[p_cellId * 27 + i];
        for(int j = 0; j < cellParticles[currentCellId]; ++j){
          if(cellOffsets[currentCellId] + j != idx){
            int potI = p_type * numPots + type[cellOffsets[currentCellId] + j];
            realG3 secPart = pos[cellOffsets[currentCellId] + j];
            // p_dist.x = __dsub_rn(p_pos.x, pos[cellOffsets[currentCellId] + j].x);
            // p_dist.y = __dsub_rn(p_pos.y, pos[cellOffsets[currentCellId] + j].y);
            // p_dist.z = __dsub_rn(p_pos.z, pos[cellOffsets[currentCellId] + j].z);
            p_dist.x = p_pos.x - secPart.x;
            p_dist.y = p_pos.y - secPart.y;
            p_dist.z = p_pos.z - secPart.z;
            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            // distSqr = 0;
            // distSqr = __fma_rn(p_dist.x, p_dist.x, distSqr);
            // distSqr = __fma_rn(p_dist.y, p_dist.y, distSqr);
            // distSqr = __fma_rn(p_dist.z, p_dist.z, distSqr);
            if(distSqr <= (s_cutoff[potI] * s_cutoff[potI])){
            // if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              // if(distSqr <= __dmul_rn(gpuPots[potI].cutoff, gpuPots[potI].cutoff)){
              if(mode == 0){
                frac2 = 1.0 / distSqr;
                // frac2 = __drcp_rn(distSqr);
                frac6 = frac2 * frac2 * frac2;
                // realG frac6 = __dmul_rn(frac2, __dmul_rn(frac2, frac2));
                calcResult = frac6 * (s_ff1[potI] * frac6 - s_ff2[potI]) * frac2;
                // calcResult = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                // calcResult = __dmul_rn(frac6, __dmul_rn((__dsub_rn(__dmul_rn(gpuPots[potI].ff1, frac6), gpuPots[potI].ff2)), frac2));
                // calcResult = __dmul_rn(frac6, __dmul_rn((__dsub_rn(__dmul_rn(s_ff1[potI], frac6), s_ff2[potI])), frac2));
                // p_force.x = __fma_rn(p_dist.x, calcResult, p_force.x);
                // p_force.y = __fma_rn(p_dist.y, calcResult, p_force.y);
                // p_force.z = __fma_rn(p_dist.z, calcResult, p_force.z);
                p_force.x += p_dist.x * calcResult;
                p_force.y += p_dist.y * calcResult;
                p_force.z += p_dist.z * calcResult;
              }
              if(mode == 1){
                frac2 = s_sigma[potI] * s_sigma[potI] / distSqr;
                // frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                frac6 = frac2 * frac2 * frac2;
                calcResult = 4.0 * s_epsilon[potI] * (frac6 * frac6 - frac6);
                // calcResult = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                p_energy += calcResult;
              }
            }
          }
        }
      }
      if(mode == 0){
        // p_force.x *= real[idx];
        // p_force.y *= real[idx];
        // p_force.z *= real[idx];

        force[idx] = p_force;
      }

      if(mode == 1){
        energy[idx] = p_energy;
      }
    }
    __global__ void 
    testKernel2( const int nPart,
                const int nCells,
                const int* id,
                const int* cellId,
                const realG3* pos,
                realG3* force,
                const realG* mass,
                const realG* drift,
                const int* type,
                const bool* real,
                const int* cellParticles, 
                const int* cellOffsets,
                const int* cellNeighbors,
                realG* energy,
                const d_LennardJonesGPU* gpuPots,
                const int numPots,
                const int mode){
      int idx = blockIdx.x * blockDim.x + threadIdx.x;
      // __shared__ realG3 s_pos[THREADSPERBLOCK];
      __shared__ realG s_pos_x[THREADSPERBLOCK];
      __shared__ realG s_pos_y[THREADSPERBLOCK];
      __shared__ realG s_pos_z[THREADSPERBLOCK];
      __shared__ int s_id[THREADSPERBLOCK];
      //__shared__ realG[THREADSPERBLOCK] s_mass;
      //__shared__ realG[THREADSPERBLOCK] s_drift;
      __shared__ int s_type[THREADSPERBLOCK];
      __shared__ int activeThreads;
      // __shared__ realG s_force_x[THREADSPERBLOCK];
      // __shared__ realG s_force_y[THREADSPERBLOCK];
      // __shared__ realG s_force_z[THREADSPERBLOCK];
      __shared__ realG s_energy[THREADSPERBLOCK];
      __shared__ int numberRuns;
      int potI;
      bool sameId;
      int calcCellOffset = cellOffsets[blockIdx.x];
      // s_force_x[threadIdx.x] = 0.0f;
      // s_force_y[threadIdx.x] = 0.0f;
      // s_force_z[threadIdx.x] = 0.0f;
      realG p_energy;

      s_energy[threadIdx.x] = 0.0f;
      p_energy = 0.0f;

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }
      if(threadIdx.x == 0){
        int numberNeighborParticles = 0;
        for(int i = 0; i < 27; ++i){
            numberNeighborParticles += cellParticles[cellNeighbors[blockIdx.x * 27 + i]];
        }
        numberRuns = (numberNeighborParticles - 1) / THREADSPERBLOCK + 1;
      }

      int currentii = 0;
      int currentjj = 0;

      __syncthreads(); 

      realG3 p_dist;
      realG3 p_force;
      realG distSqr;
      for(int i = 0; i < numberRuns; ++i){
        activeThreads = 0;
        int ii,jj;
        if(threadIdx.x == 0){
          for(ii = currentii; ii < 27; ++ii){
            // if(blockIdx.x == 41 && mode == 0) {
            //   printf("accessing neighbor cell: %d\n", cellNeighbors[blockIdx.x * 27 + ii]);
            // }
            for(jj = currentjj; jj < cellParticles[cellNeighbors[blockIdx.x * 27 + ii]]; ++jj){
              if(activeThreads == THREADSPERBLOCK){ //} || (ii == 26 && jj == cellParticles[cellNeighbors[blockIdx.x * 27 + 26]] - 1)){
                //sharedMemfull = true;
                currentii = ii;
                currentjj = jj;
                goto end;
                // if(jj == cellParticles[cellNeighbors[blockIdx.x * 27 + ii]] - 1){
                //   currentii = ii+1;
                //   currentjj = 0;
                // } else {
                //   currentjj = jj + 1;
                //   currentii = ii;
                // }
              } else{
                // if(blockIdx.x == 41 && mode == 0) {
                //   printf("activeThreads %d, ii %d, jj %d, blockIdx %d\n", activeThreads, ii, jj, blockIdx.x);
                // }
                s_pos_x[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].x;
                s_pos_y[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].y;
                s_pos_z[activeThreads] = pos[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj].z;
                s_type[activeThreads] = type[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                s_id[activeThreads] = id[cellOffsets[cellNeighbors[blockIdx.x * 27 + ii]] + jj];
                activeThreads++;
              }
            }
            currentjj = 0;
          }
          end:;
        }
        __syncthreads();
        for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
          // s_force_x[threadIdx.x] = 0.0f;
          // s_force_y[threadIdx.x] = 0.0f;
          // s_force_z[threadIdx.x] = 0.0f;
          s_energy[threadIdx.x] = 0.0f;
          p_energy = 0.0f;
          p_force.x = 0.0f;
          p_force.y = 0.0f;
          p_force.z = 0.0f;
          potI = s_type[threadIdx.x] * numPots + type[calcCellOffset + j];
          sameId = s_id[threadIdx.x] == id[calcCellOffset + j] ? true : false;
          if(threadIdx.x < activeThreads){
            //printf("threadIdx.x=%d, idx.x: %d, own particle id: %d\n", threadIdx.x, idx, s_id[threadIdx.x]);
            // p_dist.x = s_pos[threadIdx.x].x - pos[calcCellOffset + j].x;
            // p_dist.y = s_pos[threadIdx.x].y - pos[calcCellOffset + j].y;
            // p_dist.z = s_pos[threadIdx.x].z - pos[calcCellOffset + j].z;
            p_dist.x = pos[calcCellOffset + j].x - s_pos_x[threadIdx.x];
            p_dist.y = pos[calcCellOffset + j].y - s_pos_y[threadIdx.x];
            p_dist.z = pos[calcCellOffset + j].z - s_pos_z[threadIdx.x];
            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  // s_force_x[threadIdx.x] = p_dist.x * ffactor;
                  // s_force_y[threadIdx.x] = p_dist.y * ffactor;
                  // s_force_z[threadIdx.x] = p_dist.z * ffactor;
                  p_force.x = p_dist.x * ffactor;
                  p_force.y = p_dist.y * ffactor;
                  p_force.z = p_dist.z * ffactor;
                  //printf("2. id1=%d, id2=%d %f %f %f\n", id[calcCellOffset + j], s_id[threadIdx.x], p_dist.x * ffactor,  p_dist.y * ffactor, p_dist.z * ffactor);

                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                  p_energy = energy;
                  //printf("Energy threadIdx: %d, %f\n",threadIdx.x, p_energy);
                  s_energy[threadIdx.x] = energy;
                }
              }
            }
          }
          __syncthreads();
          if(mode == 0){
            p_force = blockReduceSumTriple(p_force, 0xffffffff);
            // if(threadIdx.x == 0){
            //   for(int k = 0; k < activeThreads; ++k){
            //     force[calcCellOffset + j].x += s_force_x[k];
            //     force[calcCellOffset + j].y += s_force_y[k];
            //     force[calcCellOffset + j].z += s_force_z[k];
            //   }
            // }
            // if(threadIdx.x < activeThreads){
            //   force[calcCellOffset + j].x += s_force_x[threadIdx.x];
            //   force[calcCellOffset + j].y += s_force_y[threadIdx.x];
            //   force[calcCellOffset + j].z += s_force_z[threadIdx.x];
            if(threadIdx.x == 0){
              force[calcCellOffset + j].x += p_force.x;
              force[calcCellOffset + j].y += p_force.y;
              force[calcCellOffset + j].z += p_force.z;
            }
          }
          if(mode == 1){
            p_energy = blockReduceSum(p_energy, 0xffffffff);
            if(threadIdx.x == 0){
              // energy[calcCellOffset + j] += p_energy;
              for(int k = 0; k < activeThreads; ++k){
                energy[calcCellOffset + j] += s_energy[k];
              }
              //printf("pos in memory:: %d, Energy blockIdx: %d, Penergy: %f\n", calcCellOffset + j, blockIdx.x, energy[calcCellOffset + j]);
            }
          }
          __syncthreads();
        }
        __syncthreads();
      }
    }

    __global__ void 
    testKernel3(const int nPart,
                const int nCells,
                const int* __restrict__ id,
                const int* __restrict__ cellId,
                const realG3* __restrict__ pos,
                realG3* force,
                const realG* __restrict__ mass,
                const realG* __restrict__ drift,
                const int* __restrict__ type,
                const bool* __restrict__ real,
                const int* __restrict__ cellParticles, 
                const int* __restrict__ cellOffsets,
                const int* __restrict__ cellNeighbors,
                realG* __restrict__ energy,
                const d_LennardJonesGPU* __restrict__ gpuPots,
                int numPots,
                int mode){
      // int idx = blockIdx.x * blockDim.x + threadIdx.x;
      __shared__ int numberLineParticles[9];
      __shared__ int numberLineWarps[9];

      int potI;
      bool sameId;
      int calcCellOffset = cellOffsets[blockIdx.x];

      if(cellParticles[blockIdx.x] == 0){
        return;
      }
      if(real[calcCellOffset] == false){
        return;
      }

      realG3 p_dist;
      realG3 p_force;
      realG distSqr;
      // realG t_pos_x;
      // realG t_pos_y;
      // realG t_pos_z;      
      int t_type;
      int t_id;
      realG p_energy = 0;

      int warpId = threadIdx.x / warpSize;
      int laneId = threadIdx.x % warpSize;

      int dataOffset = cellOffsets[cellNeighbors[blockIdx.x * 27 + 3 * warpId]];

      if(threadIdx.x < 9){
        numberLineParticles[threadIdx.x] = cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 0]] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 1]] +
                                            cellParticles[cellNeighbors[(blockIdx.x * 27) + (3 * threadIdx.x) + 2]];
        numberLineWarps[threadIdx.x] = (numberLineParticles[threadIdx.x ] - 1) / warpSize + 1;
      }
      // __syncthreads();
      // if(laneId == 0 && blockIdx.x == 61){
      //   printf("Mode: %d, BlockIdx.x %d, warpId: %d, numLinePart: %d, numLineWarps: %d, block Ids: %d %d %d\n",
      //       mode, blockIdx.x, warpId, numberLineParticles[warpId], numberLineWarps[warpId], cellNeighbors[(blockIdx.x * 27) + (3 * warpId)],
      //       cellNeighbors[(blockIdx.x * 27) + (3 * warpId) + 1],
      //       cellNeighbors[(blockIdx.x * 27) + (3 * warpId) + 2]);
      // }
      __syncthreads();

      for(int i = 0; i < numberLineWarps[warpId]; ++i){
        // t_pos_x = 0.0f;
        // t_pos_y = 0.0f;
        // t_pos_z = 0.0f;
        p_energy = 0.0f;
        p_force.x = 0.0f;
        p_force.y = 0.0f;
        p_force.z = 0.0f;
        // if(threadIdx.x == 0){
        //   printf("Mode: %d, This is blockIdx: %d\n", mode, blockIdx.x);
        // }
        unsigned mask = __ballot_sync(0xffffffff, i * warpSize + laneId < numberLineParticles[warpId]);
        if(i * warpSize + laneId < numberLineParticles[warpId]){
          // t_pos_x = pos[dataOffset + i * warpSize + laneId].x;
          // t_pos_y = pos[dataOffset + i * warpSize + laneId].y;
          // t_pos_z = pos[dataOffset + i * warpSize + laneId].z;
          t_type = type[dataOffset + i * warpSize + laneId];
          t_id = id[dataOffset + i * warpSize + laneId];
          for(int j = 0; j < cellParticles[blockIdx.x]; ++j){
            p_energy = 0.0f;
            p_force.x = 0.0f;
            p_force.y = 0.0f;
            p_force.z = 0.0f;
            potI = t_type * numPots + type[calcCellOffset + j];
            sameId = t_id == id[calcCellOffset + j] ? true : false;

            p_dist.x = pos[calcCellOffset + j].x - pos[dataOffset + i * warpSize + laneId].x;
            p_dist.y = pos[calcCellOffset + j].y - pos[dataOffset + i * warpSize + laneId].y;
            p_dist.z = pos[calcCellOffset + j].z - pos[dataOffset + i * warpSize + laneId].z;
            // p_dist.x = pos[calcCellOffset + j].x - t_pos_x;
            // p_dist.y = pos[calcCellOffset + j].y - t_pos_y;
            // p_dist.z = pos[calcCellOffset + j].z - t_pos_z;

            distSqr =  p_dist.x * p_dist.x + p_dist.y * p_dist.y + p_dist.z * p_dist.z;
            if(distSqr <= (gpuPots[potI].cutoff * gpuPots[potI].cutoff)){
              if(!sameId){
                if(mode == 0){
                  realG frac2 = 1.0 / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG ffactor = frac6 * (gpuPots[potI].ff1 * frac6 - gpuPots[potI].ff2) * frac2;
                  p_force.x = p_dist.x * ffactor;
                  p_force.y = p_dist.y * ffactor;
                  p_force.z = p_dist.z * ffactor;
                }
                if(mode == 1){
                  realG frac2 = gpuPots[potI].sigma * gpuPots[potI].sigma / distSqr;
                  realG frac6 = frac2 * frac2 * frac2;
                  realG energy = 4.0 * gpuPots[potI].epsilon * (frac6 * frac6 - frac6);
                  p_energy = energy;
                }
              }
            }
          
            // __syncthreads();
            //__syncwarp();
            if(mode == 0){
              p_force = warpReduceSumTriple(p_force, mask);
              // p_force = blockReduceSumTriple(p_force);
              if(laneId == 0){
              // if(threadIdx.x == 0){
                atomicAdd(&force[calcCellOffset + j].x, p_force.x);
                atomicAdd(&force[calcCellOffset + j].y, p_force.y);
                atomicAdd(&force[calcCellOffset + j].z, p_force.z);
              }
            }
            if(mode == 1){
              // p_energy = blockReduceSum(p_energy);
              p_energy = warpReduceSum(p_energy, mask);
              if(laneId == 0){
              // if(threadIdx.x == 0){
                  atomicAdd(&energy[calcCellOffset + j], p_energy);
                // energy[calcCellOffset + j] += p_energy;
              }
            }
            //__syncwarp();
            // __syncthreads();
          }
        }
        //__syncwarp();
        // __syncthreads();
      }
    }
    
  realG LJGPUdriverVl(StorageGPU* gpuStorage, d_LennardJonesGPU* gpuPots, int ptypes, int* vl, int* n_nb, int mode){
    realG *h_energy; 
    realG *d_energy;
    realG totalEnergy = 0;

    h_energy = new realG[gpuStorage->numberLocalParticles];
    hipMalloc(&d_energy, sizeof(realG) * gpuStorage->numberLocalParticles);
    hipMemset(d_energy, 0, sizeof(realG) * gpuStorage->numberLocalParticles);
    unsigned shared_mem_size = ptypes * ptypes * sizeof(realG) * 5;
    hipMemset(gpuStorage->d_force, 0, sizeof(realG3) * gpuStorage->numberLocalParticles);

    hipEvent_t start, stop;
    hipEventCreate(&start); CUERR
    hipEventCreate(&stop); CUERR
    hipEventRecord(start); CUERR
    // testKernel<<<SDIV(gpuStorage->numberLocalParticles, THREADSPERBLOCK), THREADSPERBLOCK, shared_mem_size>>>(
    //   //  testKernel2<<<gpuStorage->numberLocalCells, THREADSPERBLOCK>>>(
    //   // testKernel3<<<gpuStorage->numberLocalCells, 288>>>(
    //                         gpuStorage->numberLocalParticles, 
    //                         gpuStorage->numberLocalCells, 
    //                         gpuStorage->d_id,
    //                         gpuStorage->d_cellId,
    //                         gpuStorage->d_pos,
    //                         gpuStorage->d_force,
    //                         gpuStorage->d_mass,
    //                         gpuStorage->d_drift,
    //                         gpuStorage->d_type,
    //                         gpuStorage->d_real,
    //                         gpuStorage->d_particlesCell,
    //                         gpuStorage->d_cellOffsets,
    //                         gpuStorage->d_cellNeighbors,
    //                         d_energy,
    //                         gpuPots,
    //                         numPots,
    //                         mode
    //                       );

    verletListKernel<<<SDIV(gpuStorage->numberLocalParticles, THREADSPERBLOCK), THREADSPERBLOCK, shared_mem_size>>>(
      gpuStorage->numberLocalParticles, 
      gpuStorage->d_pos,
      gpuStorage->d_force,
      gpuStorage->d_mass,
      gpuStorage->d_drift,
      gpuStorage->d_type,
      gpuStorage->d_real,
      d_energy,
      gpuPots,
      ptypes,
      mode,
      vl,
      n_nb
    );
    hipEventRecord(stop); CUERR

    hipDeviceSynchronize(); CUERR
    hipEventSynchronize(stop); CUERR
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); CUERR
    // printf("%s kernel time: %2.6f\n", mode==0? "Force" : "Energy", milliseconds);
      if(mode == 1) {
        hipMemcpy(h_energy, d_energy, sizeof(realG) * gpuStorage->numberLocalParticles, hipMemcpyDeviceToHost); CUERR
        for (int i = 0; i < gpuStorage->numberLocalParticles; ++i){ 
          totalEnergy += h_energy[i];
        }
      }

      return totalEnergy / (double)2.0;

  }

  realG LJGPUdriver(StorageGPU* gpuStorage, d_LennardJonesGPU* gpuPots, int ptypes, int mode){
    realG *h_energy; 
    realG *d_energy;
    realG totalEnergy = 0;

    h_energy = new realG[gpuStorage->numberLocalParticles];
    hipMalloc(&d_energy, sizeof(realG) * gpuStorage->numberLocalParticles);
    hipMemset(d_energy, 0, sizeof(realG) * gpuStorage->numberLocalParticles);
    unsigned shared_mem_size = ptypes * ptypes * sizeof(realG) * 5;
    hipMemset(gpuStorage->d_force, 0, sizeof(realG3) * gpuStorage->numberLocalParticles);

    hipEvent_t start, stop;
    hipEventCreate(&start); CUERR
    hipEventCreate(&stop); CUERR
    hipEventRecord(start); CUERR
    testKernel<<<SDIV(gpuStorage->numberLocalParticles, THREADSPERBLOCK), THREADSPERBLOCK, shared_mem_size>>>(
      //  testKernel2<<<gpuStorage->numberLocalCells, THREADSPERBLOCK>>>(
      // testKernel3<<<gpuStorage->numberLocalCells, 288>>>(
                            gpuStorage->numberLocalParticles, 
                            gpuStorage->numberLocalCells, 
                            gpuStorage->d_id,
                            gpuStorage->d_cellId,
                            gpuStorage->d_pos,
                            gpuStorage->d_force,
                            gpuStorage->d_mass,
                            gpuStorage->d_drift,
                            gpuStorage->d_type,
                            gpuStorage->d_real,
                            gpuStorage->d_particlesCell,
                            gpuStorage->d_cellOffsets,
                            gpuStorage->d_cellNeighbors,
                            d_energy,
                            gpuPots,
                            ptypes,
                            mode
                          );
    
    hipEventRecord(stop); CUERR

    hipDeviceSynchronize(); CUERR
    hipEventSynchronize(stop); CUERR
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop); CUERR
    // printf("%s kernel time: %2.6f\n", mode==0? "Force" : "Energy", milliseconds);
      if(mode == 1) {
        hipMemcpy(h_energy, d_energy, sizeof(realG) * gpuStorage->numberLocalParticles, hipMemcpyDeviceToHost); CUERR
        for (int i = 0; i < gpuStorage->numberLocalParticles; ++i){ 
          totalEnergy += h_energy[i];
        }
      }

      return totalEnergy / (double)2.0;
    }
  }
}

// #endif